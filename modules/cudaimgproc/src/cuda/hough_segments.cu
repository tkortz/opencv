#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/cudaimgproc.hpp"

#include <unistd.h>

namespace cv { namespace cuda { namespace device
{
    namespace hough_segments_locks
    {
        int use_locks = 0;

        /* resource_id must be a non-negative int */
        int open_fzlp_lock(int resource_id)
        {
            if (!use_locks) return -3;

            fprintf(stdout, "[%d | %d] Attempting to open OMLP (%d) semaphore now.\n", gettid(), getpid(), OMLP_SEM);

            int lock_od = -1;
            obj_type_t protocol = OMLP_SEM;
            const char *lock_namespace = "./rtspin-locks";
            int cluster = 0;
            if (protocol >= 0) {
                /* open reference to semaphore */
                lock_od = litmus_open_lock(protocol, resource_id, lock_namespace, &cluster);
                if (lock_od < 0) {
                    perror("litmus_open_lock");
                    fprintf(stderr, "Could not open lock.\n");
                }
                else {
                    fprintf(stdout, "[%d | %d] Successfully opened OMLP semaphore lock: %d.\n", gettid(), getpid(), lock_od);
                }
            }

            return lock_od;
        }

        int lock_fzlp(int sem_od)
        {
            if (!use_locks) return -3;

            int res = -2;

            if (sem_od >= 0)
            {
                fprintf(stdout, "[%d | %d] Calling lock (%d) at time \t%llu\n", gettid(), getpid(), sem_od, litmus_clock());
                res = litmus_lock(sem_od);
                fprintf(stdout, "[%d | %d] Acquired lock at time \t%llu (status=%d)\n", gettid(), getpid(), litmus_clock(), res);
            }

            return res;
        }

        int wait_forbidden_zone(int sem_od)
        {
            if (!use_locks) return -3;

            int res = -2;

            int zone_length = 0;
            int cpu_measured = 0;

            // Just use defaults here
            zone_length = ms2ns(1); // default to 2 ms
            cpu_measured = ms2ns(3); // default to 3 ms

            if (sem_od >= 0)
            {
                fprintf(stdout, "[%d | %d] Checking FZ at time \t%llu\n", gettid(), getpid(), litmus_clock());
                res = litmus_access_forbidden_zone_check(sem_od, cpu_measured, zone_length);
                fprintf(stdout, "[%d | %d] Not in FZ at time \t%llu (status=%d)\n", gettid(), getpid(), litmus_clock(), res);
            }

            return res;
        }

        int set_fz_launch_done(int sem_od)
        {
            if (!use_locks) return -3;
            int res = -2;
            if (sem_od >= 0)
                res = litmus_set_fz_launch_done(sem_od);
            return res;
        }

        int exit_forbidden_zone(int sem_od)
        {
            if (!use_locks) return -3;

            int res = -2;

            if (sem_od >= 0)
            {
                res = litmus_exit_forbidden_zone(sem_od);
            }

            return res;
        }

        int unlock_fzlp(int sem_od)
        {
            if (!use_locks) return -3;

            int res = -2;

            if (sem_od >= 0)
            {
                fprintf(stdout, "[%d | %d] Unlocking at time \t\t%llu\n", gettid(), getpid(), litmus_clock());
                res = litmus_unlock(sem_od);
                fprintf(stdout, "[%d | %d] Unlocked at time \t\t%llu (status=%d)\n", gettid(), getpid(), litmus_clock(), res);
            }

            return res;
        }
    }

    namespace hough_segments
    {
        __device__ int g_counter;

        texture<uchar, hipTextureType2D, hipReadModeElementType> tex_mask(false, hipFilterModePoint, hipAddressModeClamp);

        __global__ void houghLinesProbabilistic(const PtrStepSzi accum,
                                                int4* out, const int maxSize,
                                                const float rho, const float theta,
                                                const int lineGap, const int lineLength,
                                                const int rows, const int cols)
        {
            const int r = blockIdx.x * blockDim.x + threadIdx.x;
            const int n = blockIdx.y * blockDim.y + threadIdx.y;

            if (r >= accum.cols - 2 || n >= accum.rows - 2)
                return;

            const int curVotes = accum(n + 1, r + 1);

            if (curVotes >= lineLength &&
                curVotes > accum(n, r) &&
                curVotes > accum(n, r + 1) &&
                curVotes > accum(n, r + 2) &&
                curVotes > accum(n + 1, r) &&
                curVotes > accum(n + 1, r + 2) &&
                curVotes > accum(n + 2, r) &&
                curVotes > accum(n + 2, r + 1) &&
                curVotes > accum(n + 2, r + 2))
            {
                const float radius = (r - (accum.cols - 2 - 1) * 0.5f) * rho;
                const float angle = n * theta;

                float cosa;
                float sina;
                sincosf(angle, &sina, &cosa);

                float2 p0 = make_float2(cosa * radius, sina * radius);
                float2 dir = make_float2(-sina, cosa);

                float2 pb[4] = {make_float2(-1, -1), make_float2(-1, -1), make_float2(-1, -1), make_float2(-1, -1)};
                float a;

                if (dir.x != 0)
                {
                    a = -p0.x / dir.x;
                    pb[0].x = 0;
                    pb[0].y = p0.y + a * dir.y;

                    a = (cols - 1 - p0.x) / dir.x;
                    pb[1].x = cols - 1;
                    pb[1].y = p0.y + a * dir.y;
                }
                if (dir.y != 0)
                {
                    a = -p0.y / dir.y;
                    pb[2].x = p0.x + a * dir.x;
                    pb[2].y = 0;

                    a = (rows - 1 - p0.y) / dir.y;
                    pb[3].x = p0.x + a * dir.x;
                    pb[3].y = rows - 1;
                }

                if (pb[0].x == 0 && (pb[0].y >= 0 && pb[0].y < rows))
                {
                    p0 = pb[0];
                    if (dir.x < 0)
                        dir = -dir;
                }
                else if (pb[1].x == cols - 1 && (pb[1].y >= 0 && pb[1].y < rows))
                {
                    p0 = pb[1];
                    if (dir.x > 0)
                        dir = -dir;
                }
                else if (pb[2].y == 0 && (pb[2].x >= 0 && pb[2].x < cols))
                {
                    p0 = pb[2];
                    if (dir.y < 0)
                        dir = -dir;
                }
                else if (pb[3].y == rows - 1 && (pb[3].x >= 0 && pb[3].x < cols))
                {
                    p0 = pb[3];
                    if (dir.y > 0)
                        dir = -dir;
                }

                float2 d;
                if (::fabsf(dir.x) > ::fabsf(dir.y))
                {
                    d.x = dir.x > 0 ? 1 : -1;
                    d.y = dir.y / ::fabsf(dir.x);
                }
                else
                {
                    d.x = dir.x / ::fabsf(dir.y);
                    d.y = dir.y > 0 ? 1 : -1;
                }

                float2 line_end[2];
                int gap;
                bool inLine = false;

                float2 p1 = p0;
                if (p1.x < 0 || p1.x >= cols || p1.y < 0 || p1.y >= rows)
                    return;

                for (;;)
                {
                    if (tex2D(tex_mask, p1.x, p1.y))
                    {
                        gap = 0;

                        if (!inLine)
                        {
                            line_end[0] = p1;
                            line_end[1] = p1;
                            inLine = true;
                        }
                        else
                        {
                            line_end[1] = p1;
                        }
                    }
                    else if (inLine)
                    {
                        if (++gap > lineGap)
                        {
                            bool good_line = ::abs(line_end[1].x - line_end[0].x) >= lineLength ||
                                             ::abs(line_end[1].y - line_end[0].y) >= lineLength;

                            if (good_line)
                            {
                                const int ind = ::atomicAdd(&g_counter, 1);
                                if (ind < maxSize)
                                    out[ind] = make_int4(line_end[0].x, line_end[0].y, line_end[1].x, line_end[1].y);
                            }

                            gap = 0;
                            inLine = false;
                        }
                    }

                    p1 = p1 + d;
                    if (p1.x < 0 || p1.x >= cols || p1.y < 0 || p1.y >= rows)
                    {
                        if (inLine)
                        {
                            bool good_line = ::abs(line_end[1].x - line_end[0].x) >= lineLength ||
                                             ::abs(line_end[1].y - line_end[0].y) >= lineLength;

                            if (good_line)
                            {
                                const int ind = ::atomicAdd(&g_counter, 1);
                                if (ind < maxSize)
                                    out[ind] = make_int4(line_end[0].x, line_end[0].y, line_end[1].x, line_end[1].y);
                            }

                        }
                        break;
                    }
                }
            }
        }

        int houghLinesProbabilistic_gpu(PtrStepSzb mask, PtrStepSzi accum, int4* out, int maxSize, float rho, float theta, int lineGap, int lineLength,
                                        const hipStream_t& stream,
                                        int omlp_sem_od = -1)
        {
            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(accum.cols - 2, block.x), divUp(accum.rows - 2, block.y));

            /* =============
             * LOCK: houghLinesProbabilistic_gpu
             */
            hough_segments_locks::lock_fzlp(omlp_sem_od);

            // Memset of counterPtr
            hough_segments_locks::wait_forbidden_zone(omlp_sem_od);
            cudaSafeCall( hipMemsetAsync(counterPtr, 0, sizeof(int), stream) );
            hough_segments_locks::set_fz_launch_done(omlp_sem_od);
            cudaSafeCall( hipStreamSynchronize(stream) );
            hough_segments_locks::exit_forbidden_zone(omlp_sem_od);

            bindTexture(&tex_mask, mask);

            // Kernel: houghLinesProbabilistic
            hough_segments_locks::wait_forbidden_zone(omlp_sem_od);
            houghLinesProbabilistic<<<grid, block, 0, stream>>>(accum,
                                                                out, maxSize,
                                                                rho, theta,
                                                                lineGap, lineLength,
                                                                mask.rows, mask.cols);
            cudaSafeCall( hipGetLastError() );
            hough_segments_locks::set_fz_launch_done(omlp_sem_od);
            cudaSafeCall( hipDeviceSynchronize() );
            hough_segments_locks::exit_forbidden_zone(omlp_sem_od);

            int totalCount;
            // Memcpy d2h of counterPtr
            hough_segments_locks::wait_forbidden_zone(omlp_sem_od);
            cudaSafeCall( hipMemcpyAsync(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost, stream) );
            hough_segments_locks::set_fz_launch_done(omlp_sem_od);
            cudaSafeCall( hipStreamSynchronize(stream) );
            hough_segments_locks::exit_forbidden_zone(omlp_sem_od);

            hough_segments_locks::unlock_fzlp(omlp_sem_od);
            /*
            * UNLOCK: houghLinesProbabilistic_gpu
            * ============= */

            totalCount = ::min(totalCount, maxSize);

            return totalCount;
        }
    }
}}}


#endif /* CUDA_DISABLER */
